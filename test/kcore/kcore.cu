#include "hip/hip_runtime.h"
#include "header.h"
#include "util.h"
#include "mapper.cuh"
#include "reducer.cuh"
#include "wtime.h"
#include "barrier.cuh"
#include "gpu_graph.cuh"
#include "meta_data.cuh"
#include "mapper_enactor.cuh"
#include "reducer_enactor.cuh"

/*user defined vertex behavior function*/
__inline__ __host__ __device__ feature_t user_mapper_push
(	vertex_t 	src,
	vertex_t	dest,
	feature_t	level,
	index_t*	beg_pos,
	weight_t	edge_weight,
	feature_t* 	vert_status,
	feature_t* 	vert_status_prev)
{
	return 1;
}

/*user defined vertex behavior function*/
__inline__ __host__ __device__ bool vertex_selector_push
(
  vertex_t vert_id, 
  feature_t level,
  vertex_t *adj_list, 
  index_t *beg_pos, 
  feature_t *vert_status,
  feature_t *vert_status_prev)
{
	if(vert_status_prev[vert_id] == vert_status[vert_id])
	{
		if(vert_status[vert_id] <= K)
		{
			vert_status_prev[vert_id] = 0;
			vert_status[vert_id] = 0;
		}
		return false;
	}
	else
	{
		if(vert_status_prev[vert_id]>K &&vert_status[vert_id]<=K)
			return true;
		else
			return false;
	}

}

/*user defined vertex behavior function*/
__inline__ __host__ __device__ feature_t user_mapper_pull
(	vertex_t 		src,
	vertex_t		dest,
	feature_t		level,
	index_t*		beg_pos,
	weight_t		edge_weight,
	feature_t* 		vert_status,
	feature_t* 		vert_status_prev)
{
	return ((vert_status_prev[src] <= K)
			&& (vert_status_prev[src]>0)); 
}

/*user defined vertex behavior function*/
//Attention, we only use pull once
// - at the beginning of kcore
__inline__ __host__ __device__ bool vertex_selector_pull
(
  vertex_t vert_id, 
  feature_t level,
  vertex_t *adj_list, 
  index_t *beg_pos, 
  feature_t *vert_status,
  feature_t *vert_status_prev)
{
	return (vert_status[vert_id] > K);
}



__device__ cb_reducer vert_selector_push_d = vertex_selector_push;
__device__ cb_reducer vert_selector_pull_d = vertex_selector_pull;
__device__ cb_mapper vert_behave_push_d = user_mapper_push;
__device__ cb_mapper vert_behave_pull_d = user_mapper_pull;

__global__ void 
init(meta_data mdata, gpu_graph ggraph)
{
	index_t tid = threadIdx.x+blockIdx.x*blockDim.x;
	
	while(tid < ggraph.vert_count)
	{
		feature_t degree = ggraph.beg_pos[tid + 1] 
			- ggraph.beg_pos[tid];
		mdata.vert_status[tid] = degree;
		mdata.vert_status_prev[tid] = degree;
	
		tid += blockDim.x*gridDim.x;
	}
}

int 
main(int args, char **argv)
{
	std::cout<<"Input: /path/to/exe /path/to/beg_pos /path/to/adj_list /path/weight_list src\n";
	if(args<5){std::cout<<"Wrong input\n";exit(-1);}
		
	double tm_map,tm_red,tm_scan;
	char *file_beg_pos = argv[1];
	char *file_adj_list = argv[2];
	char *file_weight_list = argv[3];
	vertex_t src_v = (vertex_t)atol(argv[4]);
	H_ERR(hipSetDevice(0));	
	
	//Read graph to CPU
	graph<long, long, long,vertex_t, index_t, weight_t>
	*ginst=new graph<long, long, long,vertex_t, index_t, weight_t>
	(file_beg_pos, file_adj_list, file_weight_list);
	
	cb_reducer vert_selector_push_h;
	cb_reducer vert_selector_pull_h;
	H_ERR(hipMemcpyFromSymbol(&vert_selector_push_h,HIP_SYMBOL(vert_selector_push_d),sizeof(cb_reducer)));
	H_ERR(hipMemcpyFromSymbol(&vert_selector_pull_h,HIP_SYMBOL(vert_selector_pull_d),sizeof(cb_reducer)));
	
	cb_mapper vert_behave_push_h;
	cb_mapper vert_behave_pull_h;
	H_ERR(hipMemcpyFromSymbol(&vert_behave_push_h,HIP_SYMBOL(vert_behave_push_d),sizeof(cb_reducer)));
	H_ERR(hipMemcpyFromSymbol(&vert_behave_pull_h,HIP_SYMBOL(vert_behave_pull_d),sizeof(cb_reducer)));
	
	gpu_graph ggraph(ginst);
	meta_data mdata(ginst->vert_count, ginst->edge_count);
	Barrier global_barrier(BLKS_NUM);
	mapper compute_mapper(ggraph, mdata, vert_behave_push_h, vert_behave_pull_h);
	reducer worklist_gather(ggraph, mdata, vert_selector_push_h, vert_selector_pull_h);
	H_ERR(hipDeviceSynchronize());

	init<<<256, 256>>>(mdata, ggraph);
		
	H_ERR(hipMemset(mdata.worklist_sz_sml, 0, sizeof(vertex_t)));
	H_ERR(hipMemset(mdata.worklist_sz_mid, 0, sizeof(vertex_t)));
	H_ERR(hipMemset(mdata.worklist_sz_lrg, 0, sizeof(vertex_t)));
	H_ERR(hipDeviceSynchronize());
	
	vertex_t *sml, *mid, *lrg;
	hipHostMalloc((void **)&sml, sizeof(vertex_t));
	hipHostMalloc((void **)&mid, sizeof(vertex_t));
	hipHostMalloc((void **)&lrg, sizeof(vertex_t));
	
	/*reducer*/
	double ltime = wtime();
	tm_red=wtime();
	//reducer_pull(0, ggraph, mdata, worklist_gather);
	reducer_strided_pull(0, ggraph, mdata, worklist_gather);
	tm_red=wtime()-tm_red;
	
	tm_map=wtime();
	mapper_pull(0, ggraph, mdata, compute_mapper);
	tm_map=wtime()-tm_map;
		
	int levels=0;
	
	H_ERR(hipMemcpy(sml, mdata.worklist_sz_sml, 
				sizeof(vertex_t), hipMemcpyDeviceToHost));
	H_ERR(hipMemcpy(mid, mdata.worklist_sz_mid, 
				sizeof(vertex_t), hipMemcpyDeviceToHost));
	H_ERR(hipMemcpy(lrg, mdata.worklist_sz_lrg, 
				sizeof(vertex_t), hipMemcpyDeviceToHost));
	printf("level-%d: %d (%lf, %lf)(map, reduce) seconds\n", levels, 
				sml[0]+mid[0]+lrg[0], tm_map, tm_red);
			
	
	H_ERR(hipDeviceSynchronize());
	H_ERR(hipMemset(mdata.worklist_sz_sml, 0, sizeof(vertex_t)));
	H_ERR(hipMemset(mdata.worklist_sz_mid, 0, sizeof(vertex_t)));
	H_ERR(hipMemset(mdata.worklist_sz_lrg, 0, sizeof(vertex_t)));
	H_ERR(hipDeviceSynchronize());

	levels++;
	
	for(;;levels++)
	{
		tm_red=wtime();
		reducer_push(levels, ggraph, mdata, worklist_gather);
		tm_red=wtime()-tm_red;
		
		H_ERR(hipMemcpy(sml, mdata.worklist_sz_sml, 
					sizeof(vertex_t), hipMemcpyDeviceToHost));
		H_ERR(hipMemcpy(mid, mdata.worklist_sz_mid, 
					sizeof(vertex_t), hipMemcpyDeviceToHost));
		H_ERR(hipMemcpy(lrg, mdata.worklist_sz_lrg, 
					sizeof(vertex_t), hipMemcpyDeviceToHost));

		if(sml[0] + mid[0] + lrg[0] == 0) break;

		tm_map = wtime();
		mapper_push(levels, ggraph, mdata, compute_mapper);
		tm_map = wtime() - tm_map;

		printf("level-%d: %d (%lf, %lf)(map, reduce) seconds\n", levels, 
				sml[0]+mid[0]+lrg[0], tm_map, tm_red);
			
		H_ERR(hipDeviceSynchronize());
		H_ERR(hipMemset(mdata.worklist_sz_sml, 0, sizeof(vertex_t)));
		H_ERR(hipMemset(mdata.worklist_sz_mid, 0, sizeof(vertex_t)));
		H_ERR(hipMemset(mdata.worklist_sz_lrg, 0, sizeof(vertex_t)));
		H_ERR(hipDeviceSynchronize());
	}

	ltime = wtime() - ltime;

	std::cout<<"Total time: "<<ltime<<" second(s).\n";
    
    feature_t *gpu_dist = new feature_t[ginst->vert_count];
    hipMemcpy(gpu_dist, mdata.vert_status, 
            sizeof(feature_t) * ginst->vert_count, hipMemcpyDeviceToHost);

    feature_t *cpu_dist;
    cpu_bfs<index_t, vertex_t, feature_t>
        (cpu_dist, src_v, ginst->vert_count, ginst->edge_count, ginst->beg_pos,
         ginst->adj_list);
    if (memcmp(cpu_dist, gpu_dist, sizeof(feature_t) * ginst->vert_count) == 0)
        printf("Result correct\n");
    else printf("Result wrong!\n");
}
