#include "hip/hip_runtime.h"
#include "header.h"
#include "util.h"
#include "mapper.cuh"
#include "reducer.cuh"
#include "wtime.h"
#include "barrier.cuh"
#include "gpu_graph.cuh"
#include "meta_data.cuh"
#include "mapper_enactor.cuh"
#include "reducer_enactor.cuh"

/*user defined vertex behavior function*/
__inline__ __host__ __device__ feature_t user_mapper_push
(	vertex_t 	src,
	vertex_t	dest,
	feature_t	level,
	index_t*	beg_pos,
	weight_t	edge_weight,
	feature_t* 	vert_status,
	feature_t* 	vert_status_prev)
{
	// 	if(feature_end==INFTY)
	//		return feature_src+1;
	//	else return feature_end;
	return vert_status_prev[src];
}

/*user defined vertex behavior function*/
__inline__ __host__ __device__ bool vertex_selector_push
(
  vertex_t vert_id, 
  feature_t level,
  vertex_t *adj_list, 
  index_t *beg_pos, 
  feature_t *vert_status,
  feature_t *vert_status_prev)
{
  //if(vert_status[vert_id]==level)	return true;
	//else return false;
	return true;
}

/*user defined vertex behavior function*/
__inline__ __host__ __device__ feature_t user_mapper_pull
(	vertex_t 		src,
	vertex_t		dest,
	feature_t		level,
	index_t*		beg_pos,
	weight_t		edge_weight,
	feature_t* 		vert_status,
	feature_t* 		vert_status_prev)
{
	//index_t degree=beg_pos[active_edge_src+1]-beg_pos[active_edge_src];
	return vert_status_prev[src];
		//return (feature_src==level ? feature_src+1:feature_end);
}

/*user defined vertex behavior function*/
__inline__ __host__ __device__ bool vertex_selector_pull
(
  vertex_t vert_id, 
  feature_t level,
  vertex_t *adj_list, 
  index_t *beg_pos, 
  feature_t *vert_status,
  feature_t *vert_status_prev)
{
  //if(vert_status[vert_id]==INFTY)	return true;
	//else return false;
	return true;
}



__device__ cb_reducer vert_selector_push_d = vertex_selector_push;
__device__ cb_reducer vert_selector_pull_d = vertex_selector_pull;
__device__ cb_mapper vert_behave_push_d = user_mapper_push;
__device__ cb_mapper vert_behave_pull_d = user_mapper_pull;

__global__ void 
init(meta_data mdata, gpu_graph ggraph)
{
	index_t tid = threadIdx.x+blockIdx.x*blockDim.x;
	float init_val = 1.0/ggraph.vert_count;

	while(tid < ggraph.vert_count)
	{
		mdata.vert_status[tid] = 0;
		index_t degree = ggraph.beg_pos[tid + 1] - ggraph.beg_pos[tid];
		if(degree != 0)
			mdata.vert_status_prev[tid] = init_val/degree;
		else 
			mdata.vert_status_prev[tid] = 0;
		
		tid += blockDim.x*gridDim.x;
	}
}

int 
main(int args, char **argv)
{
	std::cout<<"Input: /path/to/exe /path/to/beg_pos /path/to/adj_list /path/weight_list src\n";
	if(args<5){std::cout<<"Wrong input\n";exit(-1);}
		
	double tm_map,tm_red,tm_scan;
	char *file_beg_pos = argv[1];
	char *file_adj_list = argv[2];
	char *file_weight_list = argv[3];
	vertex_t src_v = (vertex_t)atol(argv[4]);
	H_ERR(hipSetDevice(0));	
	
	//Read graph to CPU
	graph<long, long, long,vertex_t, index_t, weight_t>
	*ginst=new graph<long, long, long,vertex_t, index_t, weight_t>
	(file_beg_pos, file_adj_list, file_weight_list);
	
	cb_reducer vert_selector_push_h;
	cb_reducer vert_selector_pull_h;
	H_ERR(hipMemcpyFromSymbol(&vert_selector_push_h,HIP_SYMBOL(vert_selector_push_d),sizeof(cb_reducer)));
	H_ERR(hipMemcpyFromSymbol(&vert_selector_pull_h,HIP_SYMBOL(vert_selector_pull_d),sizeof(cb_reducer)));
	
	cb_mapper vert_behave_push_h;
	cb_mapper vert_behave_pull_h;
	H_ERR(hipMemcpyFromSymbol(&vert_behave_push_h,HIP_SYMBOL(vert_behave_push_d),sizeof(cb_reducer)));
	H_ERR(hipMemcpyFromSymbol(&vert_behave_pull_h,HIP_SYMBOL(vert_behave_pull_d),sizeof(cb_reducer)));
	
	gpu_graph ggraph(ginst);
	meta_data mdata(ginst->vert_count, ginst->edge_count);
	Barrier global_barrier(BLKS_NUM);
	mapper compute_mapper(ggraph, mdata, vert_behave_push_h, vert_behave_pull_h);
	reducer worklist_gather(ggraph, mdata, vert_selector_push_h, vert_selector_pull_h);
	H_ERR(hipDeviceSynchronize());

	init<<<256, 256>>>(mdata, ggraph);
		
	H_ERR(hipMemset(mdata.worklist_sz_sml, 0, sizeof(vertex_t)));
	H_ERR(hipMemset(mdata.worklist_sz_mid, 0, sizeof(vertex_t)));
	H_ERR(hipMemset(mdata.worklist_sz_lrg, 0, sizeof(vertex_t)));
	H_ERR(hipDeviceSynchronize());
	
	vertex_t *sml, *mid, *lrg;
	hipHostMalloc((void **)&sml, sizeof(vertex_t));
	hipHostMalloc((void **)&mid, sizeof(vertex_t));
	hipHostMalloc((void **)&lrg, sizeof(vertex_t));
	
	/*reducer*/
	tm_red=wtime();
	reducer_pull(0, ggraph, mdata, worklist_gather);
	tm_red=wtime()-tm_red;
	
	feature_t *level, *level_h;
	hipMalloc((void **)&level, sizeof(feature_t));
	hipHostMalloc((void **)&level_h, sizeof(feature_t));
	H_ERR(hipDeviceSynchronize());
	double time=wtime();
	push_pull_opt
		(level, ggraph, mdata, compute_mapper, worklist_gather, global_barrier);
	
	//for(int levels=0;;levels++)
	//{
	//	//H_ERR(hipMemcpy(mdata.sa_chk, mdata.vert_status_prev, 
	//	//			sizeof(feature_t)*ggraph.vert_count, hipMemcpyDeviceToHost));
	//	//for(int i = 0; i < 10; i ++)
	//	//	std::cout<<mdata.sa_chk[i] * (ginst->beg_pos[i+1] - ginst->beg_pos[i])<<" ";
	//	//std::cout<<"\n";
	//	
	//	/* mapper */
	//	tm_map=wtime();
	//	mapper_pull(level, ggraph, mdata, compute_mapper);
	//	tm_map=wtime()-tm_map;
	//	
	//	feature_t *tmp = compute_mapper.vert_status;
	//	compute_mapper.vert_status = compute_mapper.vert_status_prev;
	//	compute_mapper.vert_status_prev = tmp;
	//	
	//	//H_ERR(hipMemcpy(sml, mdata.worklist_sz_sml, sizeof(vertex_t), hipMemcpyDeviceToHost));
	//	//H_ERR(hipMemcpy(mid, mdata.worklist_sz_mid, sizeof(vertex_t), hipMemcpyDeviceToHost));
	//	//H_ERR(hipMemcpy(lrg, mdata.worklist_sz_lrg, sizeof(vertex_t), hipMemcpyDeviceToHost));
	//	//
	//	//printf("level-%d: %d\n", levels, sml[0]+mid[0]+lrg[0]);
	//		
	//		
	//	/*monitoring*/
	//	std::cout<<"Level: "<<(int)levels<<" " 
	//		<<"Time (map, reduce): "<<tm_map<<" "<<tm_red<<"\n";

	//	if(levels == 10)break;
	//}
	std::cout<<"Total time: "<<wtime()-time<<" second(s).\n";
	//dumper(ggraph,mdata);
}
